#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include "vectorSum.h"

static const int n_el = 512;
static const size_t size = n_el * sizeof(float);
// declare the kernel function
__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el)
{
  // calculate the unique thread index
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  // perform tid-th elements addition 
  if (tid < n_el) C[tid] = A[tid] + B[tid];
}


// function which invokes the kernel
void sum(const float* A, const float* B, float* C, int n_el) {

  // declare the number of blocks per grid and the number of threads per block
  int threadsPerBlock,blocksPerGrid;

  // use 1 to 512 threads per block
  if (n_el<512){
    threadsPerBlock = n_el;
    blocksPerGrid   = 1;
  } else {
    threadsPerBlock = 512;
    blocksPerGrid   = ceil(double(n_el)/double(threadsPerBlock));
  }

  // invoke the kernel
  kernel_sum<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, n_el);
}

// kernel

float* myCudaTest(int *n){
	*n=n_el;
// declare and allocate input vectors h_A and h_B in the host (CPU) memory
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_C = (float*)malloc(size);

  // declare device vectors in the device (GPU) memory
  float *d_A,*d_B,*d_C;

  // initialize input vectors
  for (int i=0; i<n_el; i++){
    h_A[i]=sin(i);
    h_B[i]=cos(i);
  }

  // allocate device vectors in the device (GPU) memory
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  // copy input vectors from the host (CPU) memory to the device (GPU) memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // call kernel function
  sum(d_A, d_B, d_C, n_el);

  // copy the output (results) vector from the device (GPU) memory to the host (CPU) memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  // free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  delete[] h_A;
  delete[] h_B;
  return h_C;
}